#include "hip/hip_runtime.h"
/* Copyright (c) 2019-2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "MemoryTracker.h"

#include <sanitizer_patching.h>

static __device__
SanitizerPatchResult CommonCallback(
    void* userdata,
    uint64_t pc,
    void* ptr,
    uint32_t accessSize,
    uint32_t flags,
    MemoryAccessType type)
{
    auto* pTracker = (MemoryAccessTracker*)userdata;

    uint32_t old = atomicAdd(&(pTracker->currentEntry), 1);

    // no more space!
    if (old >= pTracker->maxEntry)
        return SANITIZER_PATCH_SUCCESS;

    MemoryAccess& access = pTracker->accesses[old];
    access.address = (uint64_t)(uintptr_t)ptr;
    access.accessSize = accessSize;
    access.flags = flags;
    access.threadId = threadIdx;
    access.type = type;

    return SANITIZER_PATCH_SUCCESS;
}

extern "C" __device__ __noinline__
SanitizerPatchResult MemoryGlobalAccessCallback(
    void* userdata,
    uint64_t pc,
    void* ptr,
    uint32_t accessSize,
    uint32_t flags)
{
    return CommonCallback(userdata, pc, ptr, accessSize, flags, MemoryAccessType::Global);
}

extern "C" __device__ __noinline__
SanitizerPatchResult MemorySharedAccessCallback(
    void* userdata,
    uint64_t pc,
    void* ptr,
    uint32_t accessSize,
    uint32_t flags)
{
    return CommonCallback(userdata, pc, ptr, accessSize, flags, MemoryAccessType::Shared);
}

extern "C" __device__ __noinline__
SanitizerPatchResult MemoryLocalAccessCallback(
    void* userdata,
    uint64_t pc,
    void* ptr,
    uint32_t accessSize,
    uint32_t flags)
{
    return CommonCallback(userdata, pc, ptr, accessSize, flags, MemoryAccessType::Local);
}

extern "C" __device__ __noinline__
SanitizerPatchResult MemcpyAsyncCallback(void* userdata, uint64_t pc, void* src, uint32_t dst, uint32_t accessSize, uint32_t totalShmemSize)
{
    if (src)
    {
        CommonCallback(userdata, pc, src, accessSize, SANITIZER_MEMORY_DEVICE_FLAG_READ, MemoryAccessType::Global);
    }

    return CommonCallback(userdata, pc, (void*)dst, accessSize, SANITIZER_MEMORY_DEVICE_FLAG_WRITE, MemoryAccessType::Shared);
}
